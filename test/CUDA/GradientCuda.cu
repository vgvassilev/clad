#include "hip/hip_runtime.h"
// The Test checks whether a clad gradient can be successfully be generated on
// the device having all the dependencies also as device functions.

// RUN: %cladclang_cuda -I%S/../../include  %s -fsyntax-only \
// RUN: %cudasmlevel --cuda-path=%cudapath  -Xclang -verify 2>&1 | FileCheck %s

// RUN: %cladclang_cuda -I%S/../../include %s -xc++ %cudasmlevel \
// RUN: --cuda-path=%cudapath -L/usr/local/cuda/lib64 -lcudart_static \
// RUN: -ldl -lrt -pthread -lm -lstdc++

// REQUIRES: cuda-runtime

// expected-no-diagnostics

// XFAIL: clang-15

#include "clad/Differentiator/Differentiator.h"
#include <array>

#define N 3

__device__ __host__ double gauss(double* x, double* p, double sigma, int dim) {
   double t = 0;
   for (int i = 0; i< dim; i++)
       t += (x[i] - p[i]) * (x[i] - p[i]);
   t = -t / (2*sigma*sigma);
   return std::pow(2*M_PI, -dim/2.0) * std::pow(sigma, -0.5) * std::exp(t);
}


// CHECK:    void gauss_grad_1(double *x, double *p, double sigma, int dim, clad::array_ref<double> _d_p) __attribute__((device)) __attribute__((host)) {
//CHECK-NEXT:     double _d_sigma = 0;
//CHECK-NEXT:     int _d_dim = 0;
//CHECK-NEXT:     double _d_t = 0;
//CHECK-NEXT:     unsigned long _t0;
//CHECK-NEXT:     int _d_i = 0;
//CHECK-NEXT:     int i = 0;
//CHECK-NEXT:     clad::tape<double> _t1 = {};
//CHECK-NEXT:     double _t2;
//CHECK-NEXT:     double _t3;
//CHECK-NEXT:     double _t4;
//CHECK-NEXT:     double _t5;
//CHECK-NEXT:     double _t6;
//CHECK-NEXT:     double t = 0;
//CHECK-NEXT:     _t0 = 0;
//CHECK-NEXT:     for (i = 0; i < dim; i++) {
//CHECK-NEXT:         _t0++;
//CHECK-NEXT:         clad::push(_t1, t);
//CHECK-NEXT:         t += (x[i] - p[i]) * (x[i] - p[i]);
//CHECK-NEXT:     }
//CHECK-NEXT:     _t2 = t;
//CHECK-NEXT:     _t3 = (2 * sigma * sigma);
//CHECK-NEXT:     t = -t / _t3;
//CHECK-NEXT:     _t6 = std::pow(2 * 3.1415926535897931, -dim / 2.);
//CHECK-NEXT:     _t5 = std::pow(sigma, -0.5);
//CHECK-NEXT:     _t4 = std::exp(t);
//CHECK-NEXT:     goto _label0;
//CHECK-NEXT:   _label0:
//CHECK-NEXT:     {
//CHECK-NEXT:         double _r1 = 0;
//CHECK-NEXT:         double _r2 = 0;
//CHECK-NEXT:         clad::custom_derivatives{{(::std)?}}::pow_pullback(2 * 3.1415926535897931, -dim / 2., 1 * _t4 * _t5, &_r1, &_r2);
//CHECK-NEXT:         _d_dim += -_r2 / 2.;
//CHECK-NEXT:         double _r3 = 0;
//CHECK-NEXT:         double _r4 = 0;
//CHECK-NEXT:         clad::custom_derivatives{{(::std)?}}::pow_pullback(sigma, -0.5, _t6 * 1 * _t4, &_r3, &_r4);
//CHECK-NEXT:         _d_sigma += _r3;
//CHECK-NEXT:         double _r5 = 0;
//CHECK-NEXT:         _r5 += _t6 * _t5 * 1 * clad::custom_derivatives::exp_pushforward(t, 1.).pushforward;
//CHECK-NEXT:         _d_t += _r5;
//CHECK-NEXT:     }
//CHECK-NEXT:     {
//CHECK-NEXT:         t = _t2;
//CHECK-NEXT:         double _r_d1 = _d_t;
//CHECK-NEXT:         _d_t -= _r_d1;
//CHECK-NEXT:         _d_t += -_r_d1 / _t3;
//CHECK-NEXT:         double _r0 = _r_d1 * --t / (_t3 * _t3);
//CHECK-NEXT:         _d_sigma += 2 * _r0 * sigma;
//CHECK-NEXT:         _d_sigma += 2 * sigma * _r0;
//CHECK-NEXT:     }
//CHECK-NEXT:     for (; _t0; _t0--) {
//CHECK-NEXT:         i--;
//CHECK-NEXT:         t = clad::pop(_t1);
//CHECK-NEXT:         double _r_d0 = _d_t;
//CHECK-NEXT:         _d_p[i] += -_r_d0 * (x[i] - p[i]);
//CHECK-NEXT:         _d_p[i] += -(x[i] - p[i]) * _r_d0;
//CHECK-NEXT:     }
//CHECK-NEXT: }

__global__ void compute(double* d_x, double* d_p, int n, double* d_result) {
  auto gauss_g = clad::gradient(gauss, "p");
  gauss_g.execute(d_x, d_p, 2.0, n, d_result);
}

int main(void) {
  double *x, *d_x;
  double *p, *d_p;

  x = (double*)malloc(N * sizeof(double));
  p = (double*)malloc(N * sizeof(double));
  for (int i = 0; i < N; i++) {
    x[i] = 2.0;
    p[i] = 1.0;
  }

  hipMalloc(&d_x, N * sizeof(double));
  hipMemcpy(d_x, x, N * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc(&d_p, N * sizeof(double));
  hipMemcpy(d_p, p, N * sizeof(double), hipMemcpyHostToDevice);
  std::array<double, N> result{0};
  double *d_result;

  hipMalloc(&d_result, N * sizeof(double));

  compute<<<1, 1>>>(d_x, d_p, N, d_result);
  hipDeviceSynchronize();

  hipMemcpy(result.data(), d_result, N * sizeof(double), hipMemcpyDeviceToHost);
  printf("%f,%f,%f\n", result[0], result[1], result[2]);

  std::array<double, N> result_cpu{0};
  auto gauss_g = clad::gradient(gauss, "p");
  gauss_g.execute(x, p, 2.0, N, result_cpu.data());

  if (result != result_cpu) {
    printf("Results are not equal\n");
    return 1;
  }

}
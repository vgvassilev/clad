#include "hip/hip_runtime.h"
// The Test checks whether a clad gradient can be successfully be generated on
// the device having all the dependencies also as device functions.

// RUN: %cladclang_cuda -I%S/../../include -fsyntax-only \
// RUN:     --cuda-gpu-arch=%cudaarch --cuda-path=%cudapath -Xclang -verify \
// RUN:     %s 2>&1 | %filecheck %s
//
// RUN: %cladclang_cuda -I%S/../../include --cuda-gpu-arch=%cudaarch \
// RUN:      --cuda-path=%cudapath %cudaldflags -oGradientCuda.out %s
//
// RUN: ./GradientCuda.out | %filecheck_exec %s
//
// REQUIRES: cuda-runtime
//
// expected-no-diagnostics

#include "clad/Differentiator/Differentiator.h"
#include <array>

#define N 3

__device__ __host__ double gauss(double* x, double* p, double sigma, int dim) {
   double t = 0;
   for (int i = 0; i< dim; i++)
       t += (x[i] - p[i]) * (x[i] - p[i]);
   t = -t / (2*sigma*sigma);
   return std::pow(2*M_PI, -dim/2.0) * std::pow(sigma, -0.5) * std::exp(t);
}


// CHECK: __attribute__((device)) __attribute__((host)) void gauss_grad_1(double *x, double *p, double sigma, int dim, double *_d_p) {
//CHECK-NEXT:     double _d_sigma = 0.;
//CHECK-NEXT:     int _d_dim = 0;
//CHECK-NEXT:     int _d_i = 0;
//CHECK-NEXT:     int i = 0;
//CHECK-NEXT:     clad::tape<double> _t1 = {};
//CHECK-NEXT:     double _d_t = 0.;
//CHECK-NEXT:     double t = 0;
//CHECK-NEXT:     unsigned long _t0 = {{0U|0UL|0ULL}};
//CHECK-NEXT:     for (i = 0; ; i++) {
//CHECK-NEXT:         {
//CHECK-NEXT:             if (!(i < dim))
//CHECK-NEXT:                 break;
//CHECK-NEXT:         }
//CHECK-NEXT:         _t0++;
//CHECK-NEXT:         clad::push(_t1, t);
//CHECK-NEXT:         t += (x[i] - p[i]) * (x[i] - p[i]);
//CHECK-NEXT:     }
//CHECK-NEXT:     double _t2 = t;
//CHECK-NEXT:     double _t3 = (2 * sigma * sigma);
//CHECK-NEXT:     t = -t / _t3;
//CHECK-NEXT:     double _t6 = std::pow(2 * 3.1415926535897931, -dim / 2.);
//CHECK-NEXT:     double _t5 = std::pow(sigma, -0.5);
//CHECK-NEXT:     double _t4 = std::exp(t);
//CHECK-NEXT:     {
//CHECK-NEXT:         double _r1 = 0.;
//CHECK-NEXT:         double _r2 = 0.;
//CHECK-NEXT:         clad::custom_derivatives{{(::std)?}}::pow_pullback(2 * 3.1415926535897931, -dim / 2., 1 * _t4 * _t5, &_r1, &_r2);
//CHECK-NEXT:         _d_dim += -_r2 / 2.;
//CHECK-NEXT:         double _r3 = 0.;
//CHECK-NEXT:         double _r4 = 0.;
//CHECK-NEXT:         clad::custom_derivatives{{(::std)?}}::pow_pullback(sigma, -0.5, _t6 * 1 * _t4, &_r3, &_r4);
//CHECK-NEXT:         _d_sigma += _r3;
//CHECK-NEXT:         double _r5 = 0.;
//CHECK-NEXT:         _r5 += _t6 * _t5 * 1 * clad::custom_derivatives::exp_pushforward(t, 1.).pushforward;
//CHECK-NEXT:         _d_t += _r5;
//CHECK-NEXT:     }
//CHECK-NEXT:     {
//CHECK-NEXT:         t = _t2;
//CHECK-NEXT:         double _r_d1 = _d_t;
//CHECK-NEXT:         _d_t = 0.;
//CHECK-NEXT:         _d_t += -_r_d1 / _t3;
//CHECK-NEXT:         double _r0 = _r_d1 * -(-t / (_t3 * _t3));
//CHECK-NEXT:         _d_sigma += 2 * _r0 * sigma;
//CHECK-NEXT:         _d_sigma += 2 * sigma * _r0;
//CHECK-NEXT:     }
//CHECK-NEXT:     for (;; _t0--) {
//CHECK-NEXT:         {
//CHECK-NEXT:             if (!_t0)
//CHECK-NEXT:                 break;
//CHECK-NEXT:         }
//CHECK-NEXT:         i--;
//CHECK-NEXT:         t = clad::pop(_t1);
//CHECK-NEXT:         double _r_d0 = _d_t;
//CHECK-NEXT:         _d_p[i] += -_r_d0 * (x[i] - p[i]);
//CHECK-NEXT:         _d_p[i] += -(x[i] - p[i]) * _r_d0;
//CHECK-NEXT:     }
//CHECK-NEXT: }

__global__ void compute(double* d_x, double* d_p, int n, double* d_result) {
  auto gauss_g = clad::gradient(gauss, "p");
  gauss_g.execute(d_x, d_p, 2.0, n, d_result);
}

int main(void) {
  double *x, *d_x;
  double *p, *d_p;

  x = (double*)malloc(N * sizeof(double));
  p = (double*)malloc(N * sizeof(double));
  for (int i = 0; i < N; i++) {
    x[i] = 2.0;
    p[i] = 1.0;
  }

  hipMalloc(&d_x, N * sizeof(double));
  hipMemcpy(d_x, x, N * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc(&d_p, N * sizeof(double));
  hipMemcpy(d_p, p, N * sizeof(double), hipMemcpyHostToDevice);
  std::array<double, N> result{0};
  double *d_result;

  hipMalloc(&d_result, N * sizeof(double));

  compute<<<1, 1>>>(d_x, d_p, N, d_result);
  hipDeviceSynchronize();

  hipMemcpy(result.data(), d_result, N * sizeof(double), hipMemcpyDeviceToHost);
  printf("%f,%f,%f\n", result[0], result[1], result[2]); // CHECK-EXEC: 0.007714,0.007714,0.007714

  std::array<double, N> result_cpu{0};
  auto gauss_g = clad::gradient(gauss, "p");
  gauss_g.execute(x, p, 2.0, N, result_cpu.data());

  if (result != result_cpu) {
    printf("Results are not equal\n");
    return 1;
  }

}

#include "hip/hip_runtime.h"
// The Test checks whether a clad gradient can be successfully be generated on
// the device having all the dependencies also as device functions.

// RUN: %cladclang_cuda -I%S/../../include  %s -fsyntax-only \
// RUN: %cudasmlevel --cuda-path=%cudapath  -Xclang -verify 2>&1 | FileCheck %s

// RUN: %cladclang_cuda -I%S/../../include %s -xc++ %cudasmlevel \
// RUN: --cuda-path=%cudapath -L/usr/local/cuda/lib64 -lcudart_static \
// RUN: -ldl -lrt -pthread -lm -lstdc++

// REQUIRES: cuda-runtime

// expected-no-diagnostics

// XFAIL: clang-15

#include "clad/Differentiator/Differentiator.h"

#define N 3

__device__ __host__ double gauss(double* x, double* p, double sigma, int dim) {
   double t = 0;
   for (int i = 0; i< dim; i++)
       t += (x[i] - p[i]) * (x[i] - p[i]);
   t = -t / (2*sigma*sigma);
   return std::pow(2*M_PI, -dim/2.0) * std::pow(sigma, -0.5) * std::exp(t);
}

auto gauss_g = clad::gradient(gauss, "p");

// CHECK:    void gauss_grad_1(double *x, double *p, double sigma, int dim, clad::array_ref<double> _d_p) __attribute__((device)) __attribute__((host)) {
//CHECK-NEXT:     double _d_sigma = 0;
//CHECK-NEXT:     int _d_dim = 0;
//CHECK-NEXT:     double _d_t = 0;
//CHECK-NEXT:     unsigned long _t0;
//CHECK-NEXT:     int _d_i = 0;
//CHECK-NEXT:     clad::tape<double> _t1 = {};
//CHECK-NEXT:     double _t2;
//CHECK-NEXT:     double _t3;
//CHECK-NEXT:     double _t4;
//CHECK-NEXT:     double _t5;
//CHECK-NEXT:     double _t6;
//CHECK-NEXT:     double t = 0;
//CHECK-NEXT:     _t0 = 0;
//CHECK-NEXT:     for (int i = 0; i < dim; i++) {
//CHECK-NEXT:         _t0++;
//CHECK-NEXT:         clad::push(_t1, t);
//CHECK-NEXT:         t += (x[i] - p[i]) * (x[i] - p[i]);
//CHECK-NEXT:     }
//CHECK-NEXT:     _t2 = t;
//CHECK-NEXT:     _t3 = (2 * sigma * sigma);
//CHECK-NEXT:     t = -t / _t3;
//CHECK-NEXT:     _t6 = 2.;
//CHECK-NEXT:     _t5 = std::pow(sigma, -0.5);
//CHECK-NEXT:     _t4 = std::exp(t);
//CHECK-NEXT:     goto _label0;
//CHECK-NEXT:   _label0:
//CHECK-NEXT:     {
//CHECK-NEXT:         double _r8 = 1 * _t4;
//CHECK-NEXT:         double _r9 = _r8 * _t5;
//CHECK-NEXT:         double _grad0 = 0.;
//CHECK-NEXT:         double _grad1 = 0.;
//CHECK-NEXT:         clad::custom_derivatives{{(::std)?}}::pow_pullback(2 * 3.1415926535897931, -dim / _t6, _r9, &_grad0, &_grad1);
//CHECK-NEXT:         double _r10 = _grad0;
//CHECK-NEXT:         double _r11 = _r10 * 3.1415926535897931;
//CHECK-NEXT:         double _r12 = _grad1;
//CHECK-NEXT:         double _r13 = _r12 / _t6;
//CHECK-NEXT:         _d_dim += -_r13;
//CHECK-NEXT:         double _r14 = _r12 * --dim / (_t6 * _t6);
//CHECK-NEXT:         double _r15 = std::pow(2 * 3.1415926535897931, -dim / _t6) * _r8;
//CHECK-NEXT:         double _grad2 = 0.;
//CHECK-NEXT:         double _grad3 = 0.;
//CHECK-NEXT:         clad::custom_derivatives{{(::std)?}}::pow_pullback(sigma, -0.5, _r15, &_grad2, &_grad3);
//CHECK-NEXT:         double _r16 = _grad2;
//CHECK-NEXT:         _d_sigma += _r16;
//CHECK-NEXT:         double _r17 = _grad3;
//CHECK-NEXT:         double _r18 = std::pow(2 * 3.1415926535897931, -dim / _t6) * _t5 * 1;
//CHECK-NEXT:         double _r19 = _r18 * clad::custom_derivatives::exp_pushforward(t, 1.).pushforward;
//CHECK-NEXT:         _d_t += _r19;
//CHECK-NEXT:     }
//CHECK-NEXT:     {
//CHECK-NEXT:         t = _t2;
//CHECK-NEXT:         double _r_d1 = _d_t;
//CHECK-NEXT:         double _r2 = _r_d1 / _t3;
//CHECK-NEXT:         _d_t += -_r2;
//CHECK-NEXT:         double _r3 = _r_d1 * --t / (_t3 * _t3);
//CHECK-NEXT:         double _r4 = _r3 * sigma;
//CHECK-NEXT:         double _r5 = _r4 * sigma;
//CHECK-NEXT:         double _r6 = 2 * _r4;
//CHECK-NEXT:         _d_sigma += _r6;
//CHECK-NEXT:         double _r7 = 2 * sigma * _r3;
//CHECK-NEXT:         _d_sigma += _r7;
//CHECK-NEXT:         _d_t -= _r_d1;
//CHECK-NEXT:     }
//CHECK-NEXT:     for (; _t0; _t0--) {
//CHECK-NEXT:         i--;
//CHECK-NEXT:         t = clad::pop(_t1);
//CHECK-NEXT:         double _r_d0 = _d_t;
//CHECK-NEXT:         _d_t += _r_d0;
//CHECK-NEXT:         double _r0 = _r_d0 * (x[i] - p[i]);
//CHECK-NEXT:         _d_p[i] += -_r0;
//CHECK-NEXT:         double _r1 = (x[i] - p[i]) * _r_d0;
//CHECK-NEXT:         _d_p[i] += -_r1;
//CHECK-NEXT:         _d_t -= _r_d0;
//CHECK-NEXT:     }
//CHECK-NEXT: }

__global__ void compute(decltype(gauss_g) grad, double* d_x, double* d_p, int n, double* d_result) {
  grad.execute(d_x, d_p, 2.0, n, d_result);
}

int main(void) {
  double *x, *d_x;
  double *p, *d_p;

  x = (double*)malloc(N * sizeof(double));
  p = (double*)malloc(N * sizeof(double));
  for (int i = 0; i < N; i++) {
    x[i] = 2.0;
    p[i] = 1.0;
  }

  hipMalloc(&d_x, N * sizeof(double));
  hipMemcpy(d_x, x, N * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc(&d_p, N * sizeof(double));
  hipMemcpy(d_p, p, N * sizeof(double), hipMemcpyHostToDevice);
  double *result, *d_result;

  result = (double*)malloc(N * sizeof(double));
  hipMalloc(&d_result, N * sizeof(double));

  compute<<<1, 1>>>(gauss_g, d_x, d_p, N, d_result);
  hipDeviceSynchronize();

  hipMemcpy(result, d_result, N * sizeof(double), hipMemcpyDeviceToHost);
  printf("%f,%f,%f\n", result[0], result[1], result[2]);
}

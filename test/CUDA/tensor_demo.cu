#include "hip/hip_runtime.h"
// RUN: %cladclang_cuda -I%S/../../include --cuda-path=%cudapath \
// RUN:     --cuda-gpu-arch=%cudaarch %cudaldflags -otensor_demo.out %s
// RUN: ./tensor_demo.out | %filecheck_exec %s
// REQUIRES: cuda-runtime


#include "clad/Differentiator/Differentiator.h"

typedef unsigned long long int size_type;

__device__ void computeStartStep(size_type& A_start, size_type& A_step, size_type& B_start, size_type& B_step, const int idx, const size_type A_dim[3], const size_type B_dim[3], const int contractDimA, const int contractDimB) {
    size_type A_a, A_b, A_c, B_d, B_e, B_f;

    switch (contractDimA) {
        case 0:
          A_b = idx / (A_dim[2] * B_dim[(contractDimB + 1) % 3] * B_dim[(contractDimB + 2) % 3]);
          A_c = (idx / (B_dim[(contractDimB + 1) % 3] * B_dim[(contractDimB + 2) % 3])) % A_dim[2];
          A_start = 0 + A_b * A_dim[2] + A_c;
          A_step = A_dim[1] * A_dim[2];
          break;
        case 1:
          A_a = idx / (A_dim[2] * B_dim[(contractDimB + 1) % 3] * B_dim[(contractDimB + 2) % 3]);
          A_c = (idx / (B_dim[(contractDimB + 1) % 3] * B_dim[(contractDimB + 2) % 3])) % A_dim[2];
          A_start = A_a * A_dim[1] * A_dim[2] + 0 + A_c;
          A_step = A_dim[2];
          break;
        case 2:
          A_a = idx / (A_dim[1] * B_dim[(contractDimB + 1) % 3] * B_dim[(contractDimB + 2) % 3]);
          A_b = (idx / (B_dim[(contractDimB + 1) % 3] * B_dim[(contractDimB + 2) % 3])) % A_dim[1];
          A_start = A_a * A_dim[1] * A_dim[2] + A_b * A_dim[2];
          A_step = 1;
          break;
    }

    switch (contractDimB) {
        case 0:
          B_e = (idx / B_dim[2]) % B_dim[1];
          B_f = idx % B_dim[2];
          B_start = 0 + B_e * B_dim[2] + B_f;
          B_step = B_dim[1] * B_dim[2];
          break;
        case 1:
          B_d = (idx / B_dim[2]) % B_dim[0];
          B_f = idx % B_dim[2];
          B_start = B_d * B_dim[2] * B_dim[1] + 0 + B_f;
          B_step = B_dim[2];
          break;
        case 2:
          B_d = (idx / B_dim[1]) % B_dim[0];
          B_e = idx % B_dim[1];
          B_start = B_d * B_dim[2] * B_dim[1] + B_e * B_dim[2];
          B_step = 1;
          break;
    }
}

__global__ void tensorContraction3D(float* C, const float *A, const float *B, const size_type *A_dim, const size_type *B_dim, const int contractDimA, const int contractDimB) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Each thread computes one element of the output tensor
    int totalElements = A_dim[(contractDimA + 1) % 3] * A_dim[(contractDimA + 2) % 3] * B_dim[(contractDimB + 1) % 3] * B_dim[(contractDimB + 2) % 3];
    if (idx < totalElements) {
      size_type A_start, B_start, A_step, B_step;
      size_type A_a, A_b, A_c, B_d, B_e, B_f;

      computeStartStep(A_start, A_step, B_start, B_step, idx, A_dim, B_dim, contractDimA, contractDimB);
    
      float sum = 0.0f;
      for (int i = 0; i < A_dim[contractDimA]; i++) { // A_dim[contractDimA] == B_dim[contractDimB]
          sum += A[A_start + (i * A_step)] * B[B_start + (i * B_step)];
      }

      C[idx] = sum;
    }
}

void launchTensorContraction3D(float* C, const float* A, const float* B, const size_type D1, const size_type D2, const size_type D3, const size_type D4, const size_type D5) {
    float *d_A = nullptr, *d_B = nullptr, *d_C = nullptr;

    const size_type A_size = D1 * D2 * D3 * sizeof(float);
    const size_type B_size = D3 * D4 * D5 * sizeof(float);
    const size_type C_size = D1 * D2 * D4 * D5 * sizeof(float);

    // Allocate device memory and copy data from host to device
    hipMalloc(&d_A, A_size);
    hipMalloc(&d_B, B_size);
    hipMalloc(&d_C, C_size);
    hipMemcpy(d_A, A, A_size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, B_size, hipMemcpyHostToDevice);

    const size_type A_dim[3] = {D1, D2, D3};
    const size_type B_dim[3] = {D3, D4, D5};

    size_type *d_A_dim = nullptr, *d_B_dim = nullptr;
    hipMalloc(&d_A_dim, 3 * sizeof(size_type));
    hipMalloc(&d_B_dim, 3 * sizeof(size_type));
    hipMemcpy(d_A_dim, A_dim, 3 * sizeof(size_type), hipMemcpyHostToDevice);
    hipMemcpy(d_B_dim, B_dim, 3 * sizeof(size_type), hipMemcpyHostToDevice);

    // Launch the kernel
    tensorContraction3D<<<1, 256>>>(d_C, d_A, d_B, d_A_dim, d_B_dim, /*contractDimA=*/2, /*contractDimB=*/0);

    // Copy the result from device to host
    hipMemcpy(C, d_C, C_size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_A_dim);
    hipFree(d_B_dim);
}

int main() {
    const size_type D1 = 2, D2 = 3, D3 = 4, D4 = 3, D5 = 2;
    
    float A[D1][D2][D3] = {
        {{1, 2, 3, 4}, {5, 6, 7, 8}, {9, 10, 11, 12}},
        {{13, 14, 15, 16}, {17, 18, 19, 20}, {21, 22, 23, 24}}
    };

    float B[D3][D4][D5] = {
        {{1, 2}, {3, 4}, {5, 6}},
        {{7, 8}, {9, 10}, {11, 12}},
        {{13, 14}, {15, 16}, {17, 18}},
        {{19, 20}, {21, 22}, {23, 24}}
    };

    float C[D1][D2][D4][D5] = {0};  // Result tensor

    launchTensorContraction3D(&C[0][0][0][0], &A[0][0][0], &B[0][0][0], D1, D2, D3, D4, D5);

    // Compute the gradient
    auto tensor_grad = clad::gradient(launchTensorContraction3D, "C, A, B");

    // Initialize the gradient inputs
    float gradC[D1][D2][D4][D5] = {
        {
            { {1, 1}, {1, 1}, {1, 1} }, 
            { {1, 1}, {1, 1}, {1, 1} },
            { {1, 1}, {1, 1}, {1, 1} }
        },
        {
            { {1, 1}, {1, 1}, {1, 1} },
            { {1, 1}, {1, 1}, {1, 1} },
            { {1, 1}, {1, 1}, {1, 1} }
        }
    };
    float gradA[D1][D2][D3] = {0};
    float gradB[D3][D4][D5] = {0};

    // Execute tensor contraction and its gradient
    tensor_grad.execute(&C[0][0][0][0], &A[0][0][0], &B[0][0][0], D1, D2, D3, D4, D5, &gradC[0][0][0][0], &gradA[0][0][0], &gradB[0][0][0]);

    // Print the result
    std::cout << "Result C:\n";
    for (int i = 0; i < D1; ++i) {
        for (int j = 0; j < D2; ++j) {
            for (int k = 0; k < D4; ++k) {
                for (int l = 0; l < D5; ++l) {
                    std::cout << C[i][j][k][l] << " ";
                }
                std::cout << "\n";
            }
            std::cout << "\n";
        }
        std::cout << "\n";
    }
    
    std::cout << "Result C_grad w.r.t. A:\n";
    for (int i = 0; i < D1; ++i) {
        for (int j = 0; j < D2; ++j) {
            for (int k = 0; k < D3; ++k) {
                std::cout << gradA[i][j][k] << " ";
            }
            std::cout << "\n";
        }
        std::cout << "\n";
    }

    std::cout << "Result C_grad w.r.t. B:\n";
    for (int i = 0; i < D3; ++i) {
        for (int j = 0; j < D4; ++j) {
            for (int k = 0; k < D5; ++k) {
                std::cout << gradB[i][j][k] << " ";
            }
            std::cout << "\n";
        }
        std::cout << "\n";
    }

    return 0;
}

// CHECK-EXEC: Result C:
// CHECK-NEXT: 130 140 
// CHECK-NEXT: 150 160 
// CHECK-NEXT: 170 180 
// CHECK-NEXT:
// CHECK-NEXT: 290 316 
// CHECK-NEXT: 342 368 
// CHECK-NEXT: 394 420 
// CHECK-NEXT:
// CHECK-NEXT: 450 492 
// CHECK-NEXT: 534 576 
// CHECK-NEXT: 618 660 
// CHECK-NEXT:
// CHECK-NEXT:
// CHECK-NEXT: 610 668 
// CHECK-NEXT: 726 784 
// CHECK-NEXT: 842 900 
// CHECK-NEXT:
// CHECK-NEXT: 770 844 
// CHECK-NEXT: 918 992 
// CHECK-NEXT: 1066 1140 
// CHECK-NEXT:
// CHECK-NEXT: 930 1020 
// CHECK-NEXT: 1110 1200 
// CHECK-NEXT: 1290 1380 

// CHECK-EXEC: Result C_grad w.r.t. A:
// CHECK-NEXT: 21 57 93 129 
// CHECK-NEXT: 21 57 93 129 
// CHECK-NEXT: 21 57 93 129 
// CHECK-NEXT:
// CHECK-NEXT: 21 57 93 129 
// CHECK-NEXT: 21 57 93 129 
// CHECK-NEXT: 21 57 93 129 
// CHECK-NEXT:
// CHECK-EXEC: Result C_grad w.r.t. B:
// CHECK-NEXT: 66 66 
// CHECK-NEXT: 66 66 
// CHECK-NEXT: 66 66 
// CHECK-NEXT:
// CHECK-NEXT: 72 72 
// CHECK-NEXT: 72 72 
// CHECK-NEXT: 72 72 
// CHECK-NEXT:
// CHECK-NEXT: 78 78 
// CHECK-NEXT: 78 78 
// CHECK-NEXT: 78 78 
// CHECK-NEXT:
// CHECK-NEXT: 84 84 
// CHECK-NEXT: 84 84 
// CHECK-NEXT: 84 84 
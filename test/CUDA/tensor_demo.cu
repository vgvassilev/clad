#include "hip/hip_runtime.h"
#include "clad/Differentiator/Differentiator.h"

typedef unsigned long long int size_type;

__device__ void computeStartStep(size_type& A_start, size_type& A_step, size_type& B_start, size_type& B_step, const int idx, const size_type A_dim[3], const size_type B_dim[3], const int contractDimA, const int contractDimB) {
    size_type A_a, A_b, A_c, B_d, B_e, B_f;

    switch (contractDimA) {
        case 0:
          A_b = idx / (A_dim[2] * B_dim[(contractDimB + 1) % 3] * B_dim[(contractDimB + 2) % 3]);
          // size_type A_c = ((idx - (A_b * (A_dim[2] * B_dim[(contractDimB + 1) % 3] * B_dim[(contractDimB + 2) % 3]))) 
                          // / (B_dim[(contractDimB + 1) % 3] * B_dim[(contractDimB + 2) % 3]));
          A_c = (idx / (B_dim[(contractDimB + 1) % 3] * B_dim[(contractDimB + 2) % 3])) % A_dim[2];
          A_start = 0 + A_b * A_dim[2] + A_c;
          A_step = A_dim[1] * A_dim[2];
          break;
        case 1:
          A_a = idx / (A_dim[2] * B_dim[(contractDimB + 1) % 3] * B_dim[(contractDimB + 2) % 3]);
          // size_type A_c = ((idx - (A_a * (A_dim[2] * B_dim[(contractDimB + 1) % 3] * B_dim[(contractDimB + 2) % 3]))) 
                          // / (B_dim[(contractDimB + 1) % 3] * B_dim[(contractDimB + 2) % 3]));
          A_c = (idx / (B_dim[(contractDimB + 1) % 3] * B_dim[(contractDimB + 2) % 3])) % A_dim[2];
          A_start = A_a * A_dim[1] * A_dim[2] + 0 + A_c;
          A_step = A_dim[2];
          break;
        case 2:
          A_a = idx / (A_dim[1] * B_dim[(contractDimB + 1) % 3] * B_dim[(contractDimB + 2) % 3]);
          // size_type A_b = ((idx - (A_a * (A_dim[2] * B_dim[(contractDimB + 1) % 3] * B_dim[(contractDimB + 2) % 3]))) 
                          // / (B_dim[(contractDimB + 1) % 3] * B_dim[(contractDimB + 2) % 3]));
          A_b = (idx / (B_dim[(contractDimB + 1) % 3] * B_dim[(contractDimB + 2) % 3])) % A_dim[1];
          A_start = A_a * A_dim[1] * A_dim[2] + A_b * A_dim[2];
          A_step = 1;
          break;
    }

    switch (contractDimB) {
        case 0:
          B_e = (idx / B_dim[2]) % B_dim[1];
          B_f = idx % B_dim[2];
          B_start = 0 + B_e * B_dim[2] + B_f;
          B_step = B_dim[1] * B_dim[2];
          break;
        case 1:
          B_d = (idx / B_dim[2]) % B_dim[0];
          B_f = idx % B_dim[2];
          B_start = B_d * B_dim[2] * B_dim[1] + 0 + B_f;
          B_step = B_dim[2];
          break;
        case 2:
          B_d = (idx / B_dim[1]) % B_dim[0];
          B_e = idx % B_dim[1];
          B_start = B_d * B_dim[2] * B_dim[1] + B_e * B_dim[2];
          B_step = 1;
          break;
    }
}

__global__ void tensorContraction3D(float* C, const float *A, const float *B, const size_type *A_dim, const size_type *B_dim, const int contractDimA, const int contractDimB) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Each thread computes one element of the output tensor
    int totalElements = A_dim[(contractDimA + 1) % 3] * A_dim[(contractDimA + 2) % 3] * B_dim[(contractDimB + 1) % 3] * B_dim[(contractDimB + 2) % 3];
    if (idx < totalElements) {
      size_type A_start, B_start, A_step, B_step;
      size_type A_a, A_b, A_c, B_d, B_e, B_f;

      computeStartStep(A_start, A_step, B_start, B_step, idx, A_dim, B_dim, contractDimA, contractDimB);
    
      float sum = 0.0f;
      for (int i = 0; i < A_dim[contractDimA]; i++) { // A_dim[contractDimA] == B_dim[contractDimB]
          sum += A[A_start + (i * A_step)] * B[B_start + (i * B_step)];
      }

      C[idx] = sum;
    }
}

void launchTensorContraction3D(float* C, const float* A, const float* B, size_type D1, size_type D2, size_type D3, size_type D4, size_type D5) {
    float *d_A = nullptr, *d_B = nullptr, *d_C = nullptr;

    size_type A_size = D1 * D2 * D3 * sizeof(float);
    size_type B_size = D3 * D4 * D5 * sizeof(float);
    size_type C_size = D1 * D2 * D4 * D5 * sizeof(float);

    // Allocate device memory
    hipMalloc(&d_A, A_size);
    hipMalloc(&d_B, B_size);
    hipMalloc(&d_C, C_size);

    // Copy data from host to device
    hipMemcpy(d_A, A, A_size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, B_size, hipMemcpyHostToDevice);

    size_type A_dim[3] = {D1, D2, D3};
    size_type B_dim[3] = {D3, D4, D5};

    size_type *d_A_dim = nullptr, *d_B_dim = nullptr;
    hipMalloc(&d_A_dim, 3 * sizeof(size_type));
    hipMalloc(&d_B_dim, 3 * sizeof(size_type));
    hipMemcpy(d_A_dim, A_dim, 3 * sizeof(size_type), hipMemcpyHostToDevice);
    hipMemcpy(d_B_dim, B_dim, 3 * sizeof(size_type), hipMemcpyHostToDevice);

    // Launch the kernel
    tensorContraction3D<<<1, 256>>>(d_C, d_A, d_B, d_A_dim, d_B_dim, 2, 0);

    // Copy the result from device to host
    hipMemcpy(C, d_C, C_size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    const size_type D1 = 2, D2 = 3, D3 = 4, D4 = 3, D5 = 2;
    
    float A[D1][D2][D3] = {
        {{1, 2, 3, 4}, {5, 6, 7, 8}, {9, 10, 11, 12}},
        {{13, 14, 15, 16}, {17, 18, 19, 20}, {21, 22, 23, 24}}
    };

    float B[D3][D4][D5] = {
        {{1, 2}, {3, 4}, {5, 6}},
        {{7, 8}, {9, 10}, {11, 12}},
        {{13, 14}, {15, 16}, {17, 18}},
        {{19, 20}, {21, 22}, {23, 24}}
    };

    float C[D1][D2][D4][D5] = {0};  // Result tensor

    launchTensorContraction3D(&C[0][0][0][0], &A[0][0][0], &B[0][0][0], D1, D2, D3, D4, D5);

    auto tensor_grad = clad::gradient(launchTensorContraction3D, "C, A, B");

    // Print the result
    std::cout << "Result C:\n";
    for (int i = 0; i < D1; ++i) {
        for (int j = 0; j < D2; ++j) {
            for (int k = 0; k < D4; ++k) {
                for (int l = 0; l < D5; ++l) {
                    std::cout << C[i][j][k][l] << " ";
                }
                std::cout << "\n";
            }
            std::cout << "\n";
        }
        std::cout << "\n";
    }

    return 0;
}
